#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
	const int hole_h, const int hole_w,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i * hole_h;
        int w = w_in + j * hole_w;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[(i * hole_h) * width + j * hole_w ] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
	const int hole_h, const int hole_w,
    Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  const int kernel_h_eff = kernel_h + (kernel_h - 1) * (hole_h -1);
  const int kernel_w_eff = kernel_w + (kernel_w - 1) * (hole_w -1);
  int height_col = (height + 2 * pad_h - kernel_h_eff) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w_eff) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, height_col,
      width_col, hole_h, hole_w, data_col);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int hole_h, const int hole_w,
    float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int hole_h, const int hole_w,
    double* data_col);

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels,
    const int patch_h, const int patch_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
	const int hole_h, const int hole_w,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
	int w = index % width_col;
    int h = (index/width_col) % height_col;
    int c_im = (index / width_col / height_col) % channels;
    int h_im = h * stride_h - pad_h;
    int w_im = w * stride_w - pad_w;
    Dtype* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_im) * width + w_im;
    int c = c_im * patch_h * patch_w;
    const Dtype* data_col_ptr = data_col;
    data_col_ptr += (c * height_col + h) * width_col + w;
    for (int i = 0; i < patch_h; ++i) {
      for (int j = 0; j < patch_w; ++j) {
        int hh = h_im + i * hole_h;
        int ww = w_im + j * hole_w;
		if (hh >= 0 && hh < height && ww >= 0 && ww < width) {
			caffe_gpu_atomic_add(*data_col_ptr, &data_im_ptr[(i * hole_h) * width + j * hole_w] );
		}
		data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int hole_h, const int hole_w, Dtype* data_im) {
  const int kernel_h_eff = patch_h + (patch_h - 1) * (hole_h -1);
  const int kernel_w_eff = patch_w + (patch_w - 1) * (hole_w -1);
  int height_col = (height + 2 * pad_h - kernel_h_eff) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w_eff) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  caffe_gpu_set(channels * height * width, Dtype(0), data_im);
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, patch_h, patch_w,
      pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, hole_h, hole_w, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int hole_h, const int hole_w, float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int hole_h, const int hole_w, double* data_im);

}  // namespace caffe
