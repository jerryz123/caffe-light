#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          const bool has_attention_net_, const int attention_net_,
          Dtype* counts, const Dtype* weight) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_attention_net_ && label_value == attention_net_) {
      loss[index] = 0;
      counts[index] = 0;
	} else if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = - weight[label_value] * log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = weight[label_value];
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  const Dtype* weight = loss_weights_.gpu_data();
  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, has_attention_net_, attention_net_, counts, weight);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  if(has_attention_net_ && (this->phase_==TEST) ) {
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);
	total_count_ += count;
	total_loss_  += loss;
    // only normalize case !
    if(total_count_==0) top[0]->mutable_cpu_data()[0] = 0;
    else                top[0]->mutable_cpu_data()[0] = total_loss_/total_count_;
  } else {
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(nthreads, counts, &count);
	  if( count == Dtype(0) ) count = Dtype(1);
      loss /= count;
    } else {
      loss /= outer_num_;
    }
    top[0]->mutable_cpu_data()[0] = loss;
    if (top.size() == 2) {
      top[1]->ShareData(prob_);
    }
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, const bool has_attention_net_, const int attention_net_,
		  Dtype* counts, const Dtype* weight) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_attention_net_ && label_value == attention_net_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
	} else if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] *= weight[label_value];
	  }
      counts[index] = weight[label_value];
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
	const Dtype* weight = loss_weights_.gpu_data();
    SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, has_attention_net_, attention_net_, counts, weight);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(nthreads, counts, &count);
      if( count == Dtype(0) ) count = Dtype(1);
      caffe_gpu_scal(prob_.count(), loss_weight / count, bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
